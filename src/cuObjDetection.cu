#include "hip/hip_runtime.h"
/*
 ************************************************************************************
 *   Copyright (c) 2020 Dortmund University of Applied Sciences and Arts and others.
 *
 *   Contributors:
 *        Dortmund University of Applied Sciences and Arts -
 *        initial API and implementation
 ************************************************************************************
 * cuObjDetection.cu
 *
 *  Created on: Apr 17, 2020
 *      Author: Anand Prakash
 */

extern "C"{
    #include "mbseCuda.h"
}

/**
 * CUDA Kernel Device code
 *
 * Runnable to Process the image to detect and classify the objects by creating the Boundary Box.
 */
__global__ void
processImage(int *hostBbox, int *devBbox, int *hostImage, int *devImage, int numElements)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int sampleSize = 1024;

    if (index < numElements)
    {
        hostBbox[index] = sampleSize + 1;
        devBbox[index] = sampleSize + 1;
        hostImage[index] = sampleSize + 1;
        devImage[index] = sampleSize + 1;
    }
}

/* Runnable Host to device call to copy input data from host memory to device memory */
extern "C"
static void cudaCopyHostToDevice(int *bboxHost, int *bboxDevice, int *imageHost, int *imageDevice,
        int *devBboxHost, int *devBboxDevice, int *devImageHost, int *devImageDevice, size_t size)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    /* Copy the host input vectors to device */
    err = hipMemcpy(devBboxHost, bboxHost, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(devBboxDevice, bboxDevice, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(devImageHost, imageHost, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(devImageDevice, imageDevice, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return;
}

/* Runnable device to host call to copy output data from device memory to host memory */
extern "C"
static void cudaCopyDeviceToHost(int *bboxHost, int *bboxDevice, int *imageHost, int *imageDevice,
        int *devBboxHost, int *devBboxDevice, int *devImageHost, int *devImageDevice, size_t size)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Copy the device result vector in device memory to the host memory
    err = hipMemcpy(bboxHost, devBboxHost, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host memory
    err = hipMemcpy(bboxDevice, devBboxDevice, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host memory
    err = hipMemcpy(imageHost, devImageHost, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host memory
    err = hipMemcpy(imageDevice, devImageDevice, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return;
}


/* Function to detect the object and process the image. The output of this function is provided
 * to the pathPlanner for further processing */

extern "C"
void cuDetectObject(const char *func, detectObject *objdetected)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Take some random number of elements 512KB considered.
    unsigned int numElements = (512 * 1024);
    size_t size = numElements * sizeof(int);

    // Allocate the host input vector bboxHost
    int *bboxHost = (int *)malloc(size);
    // Allocate the host input vector bboxDevice
    int *bboxDevice = (int *)malloc(size);
    // Allocate the host output vector imageHost
    int *imageHost = (int *)malloc(size);
    // Allocate the host output vector imageDevice
    int *imageDevice = (int *)malloc(size);

    // Verify that allocations succeeded
    if (bboxHost == NULL || bboxDevice == NULL ||
            imageHost == NULL || imageDevice == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        bboxHost[i] = objdetected->bboxHostDetection;
        bboxDevice[i] = objdetected->bboxDeviceDetection;
        imageHost[i] = objdetected->imageHostDetection;
        imageDevice[i] = objdetected->imageDeviceDetection;
    }

    // Allocate the device input vector devBboxDevice
    int *devBboxHost = NULL;
    err = hipMalloc((void **)&devBboxHost, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector devBboxHost (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector devBboxDevice
    int *devBboxDevice = NULL;
    err = hipMalloc((void **)&devBboxDevice, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector devBboxDevice (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector devImageHost
    int *devImageHost = NULL;
    err = hipMalloc((void **)&devImageHost, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector devImageHost (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Allocate the device output vector devImageDevice
    int *devImageDevice = NULL;
    err = hipMalloc((void **)&devImageDevice, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector devImageDevice (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    /* Runnable Host to device call to copy input data from host memory to device memory */
    cudaCopyHostToDevice(bboxHost, bboxDevice, imageHost, imageDevice,
                            devBboxHost, devBboxDevice, devImageHost, devImageDevice, size);


    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    //fprintf(stdout,"CUDA kernel launch with %d blocks of %d threads from parent thread %s\n"
    //        , blocksPerGrid, threadsPerBlock, func);
    /* Runnable to process the image */
    processImage<<<blocksPerGrid, threadsPerBlock>>>(devBboxHost, devBboxDevice, devImageHost, devImageDevice, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /* Runnable device to host call to copy output data from device memory to host memory */
    cudaCopyDeviceToHost(bboxHost, bboxDevice, imageHost, imageDevice,
            devBboxHost, devBboxDevice, devImageHost, devImageDevice, size);

    /* Copy the data to output buffer */
    memcpy((int *)&objdetected->bboxDeviceDetection, (int *)&bboxDevice[0], sizeof(int));
    memcpy((int *)&objdetected->bboxHostDetection, (int *)&bboxHost[0], sizeof(int));
    memcpy((int *)&objdetected->imageDeviceDetection, (int *)&imageDevice[0], sizeof(int));
    memcpy((int *)&objdetected->imageHostDetection, (int *)&imageHost[0], sizeof(int));

    // Free device global memory
    err = hipFree(devBboxDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(devBboxHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(devImageDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(devImageHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(bboxDevice);
    free(imageDevice);
    free(bboxHost);
    free(imageHost);

    bboxDevice = bboxHost = imageDevice = imageHost = NULL;

    printf("Detection task is complete.\n");
}

